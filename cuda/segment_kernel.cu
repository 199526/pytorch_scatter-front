#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>

#include "atomics.cuh"
#include "compat.cuh"

#define THREADS 256
#define BLOCKS(TB, N) (TB * N + THREADS - 1) / THREADS
#define FULL_MASK 0xffffffff

enum ReductionType { ADD, MEAN, MIN, MAX };

#define AT_DISPATCH_REDUCTION_TYPES(reduce, ...)                               \
  [&] {                                                                        \
    if (reduce == "add") {                                                     \
      const ReductionType REDUCE = ADD;                                        \
      return __VA_ARGS__();                                                    \
    } else if (reduce == "mean") {                                             \
      const ReductionType REDUCE = MEAN;                                       \
      return __VA_ARGS__();                                                    \
    } else if (reduce == "min") {                                              \
      const ReductionType REDUCE = MIN;                                        \
      return __VA_ARGS__();                                                    \
    } else if (reduce == "max") {                                              \
      const ReductionType REDUCE = MAX;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
  }()

template <typename scalar_t, ReductionType REDUCE> struct Reducer {
  static inline __host__ __device__ scalar_t init() {
    if (REDUCE == MIN) {
      return std::numeric_limits<scalar_t>::max();
    } else if (REDUCE == MAX) {
      return std::numeric_limits<scalar_t>::min();
    } else {
      return (scalar_t)0;
    }
  }

  static inline __host__ __device__ void update(scalar_t *val, scalar_t new_val,
                                                int64_t *arg, int64_t new_arg) {
    if ((REDUCE == MIN && new_val < *val) ||
        (REDUCE == MAX && new_val > *val)) {
      *val = new_val;
      *arg = new_arg;
    } else {
      *val = *val + new_val;
    }
  }

  static inline __host__ __device__ void write(scalar_t *address, scalar_t val,
                                               int64_t *arg_address,
                                               int64_t arg, int count) {
    if (REDUCE == ADD) {
      *address = val;
    } else if (REDUCE == MEAN) {
      *address = val / (scalar_t)max(count, 1);
    } else if (REDUCE == MIN || REDUCE == MAX) {
      if (count > 0) {
        *address = val;
        *arg_address = arg;
      } else {
        *address = (scalar_t)0;
      }
    }
  }

  static inline __device__ void atom_write(scalar_t *address, scalar_t val,
                                           int64_t *arg_address, int64_t arg) {
    if (REDUCE == ADD) {
      atomAdd(address, val);
    } else if (REDUCE == MEAN) {
      atomAdd(address, val);
    } else if (REDUCE == MIN && val < *address) {
      atomMin(address, val);
    } else if (REDUCE == MAX && val > *address) {
      atomMax(address, val);
    }

    if (REDUCE == MIN || REDUCE == MAX) {
      __syncthreads();
      if (*address == val) {
        *arg_address = arg;
      }
    }
  }
};

// We need our own `IndexToOffset` implementation since we do not want to
// access the last element of the `indexptr`.
template <typename scalar_t> struct IndexPtrToOffset {
  static inline __host__ __device__ int
  get(int idx, const at::cuda::detail::TensorInfo<scalar_t, int> &info) {
    int offset = idx % (info.sizes[info.dims - 1] - 1);
    offset *= info.strides[info.dims - 1];
    idx /= info.sizes[info.dims - 1] - 1;
    for (int i = info.dims - 2; i >= 0; --i) {
      offset += (idx % info.sizes[i]) * info.strides[i];
      idx /= info.sizes[i];
    }
    return offset;
  }
};

template <typename scalar_t, ReductionType REDUCE, int TB>
__global__ void
segment_csr_kernel(const scalar_t *src_data,
                   const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
                   scalar_t *out_data, int64_t *arg_out_data, size_t N,
                   size_t E) {

  // Each warp processes exactly `32/TB` rows and aggregates all row values
  // via a parallel reduction.

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / TB;
  int lane_idx = thread_idx & (TB - 1);

  if (row_idx < N) {
    int offset = IndexPtrToOffset<int64_t>::get(row_idx, indptr_info);
    int row_start = __ldg(indptr_info.data + offset);
    int row_end = __ldg(indptr_info.data + offset +
                        indptr_info.strides[indptr_info.dims - 1]);

    scalar_t val = Reducer<scalar_t, REDUCE>::init();
    int64_t arg, arg_tmp;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E;
    for (int src_idx = row_start + lane_idx; src_idx < row_end; src_idx += TB) {
      Reducer<scalar_t, REDUCE>::update(&val, src_data[offset + src_idx], &arg,
                                        src_idx);
    }

#pragma unroll
    for (int i = TB / 2; i > 0; i /= 2) {
      // Parallel reduction inside a single warp.
      if (REDUCE == MIN || REDUCE == MAX) {
        arg_tmp = __shfl_down_sync(FULL_MASK, arg, i);
      }
      Reducer<scalar_t, REDUCE>::update(
          &val, __shfl_down_sync(FULL_MASK, val, i), &arg, arg_tmp);
    }

    if (lane_idx == 0) {
      Reducer<scalar_t, REDUCE>::write(out_data + row_idx, val,
                                       arg_out_data + row_idx, arg,
                                       row_end - row_start);
    }
  }
}

template <typename scalar_t, ReductionType REDUCE>
__global__ void segment_csr_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
    scalar_t *out_data, int64_t *arg_out_data, size_t N, size_t K, size_t E) {

  // Each thread processes exactly one row. It turned out that is more
  // efficient than using shared memory due to avoiding synchronization
  // barriers.

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / K;
  int lane_idx = thread_idx % K;

  if (thread_idx < N * K) {
    int offset = IndexPtrToOffset<int64_t>::get(row_idx, indptr_info);
    int row_start = __ldg(indptr_info.data + offset);
    int row_end = __ldg(indptr_info.data + offset +
                        indptr_info.strides[indptr_info.dims - 1]);

    scalar_t val = Reducer<scalar_t, REDUCE>::init();
    int64_t arg;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E * K;
    for (int src_idx = row_start; src_idx < row_end; src_idx++) {
      Reducer<scalar_t, REDUCE>::update(
          &val, src_data[offset + K * src_idx + lane_idx], &arg, src_idx);
    }

    Reducer<scalar_t, REDUCE>::write(out_data + thread_idx, val,
                                     arg_out_data + thread_idx, arg,
                                     row_end - row_start);
  }
}

std::tuple<at::Tensor, at::optional<at::Tensor>>
segment_csr_cuda(at::Tensor src, at::Tensor indptr,
                 at::optional<at::Tensor> out_opt, std::string reduce) {

  AT_ASSERTM(src.dim() >= indptr.dim());
  for (int i = 0; i < indptr.dim() - 1; i++)
    AT_ASSERTM(src.size(i) == indptr.size(i));

  src = src.contiguous();
  auto reduce_dim = indptr.dim() - 1;

  at::Tensor out;
  if (out_opt.has_value()) {
    out = out_opt.value().contiguous();
    for (int i = 0; i < out.dim(); i++)
      if (i != reduce_dim)
        AT_ASSERTM(src.size(i) == out.size(i));
    AT_ASSERTM(out.size(reduce_dim) == indptr.size(reduce_dim) - 1);
  } else {
    auto sizes = src.sizes().vec();
    sizes[reduce_dim] = indptr.size(reduce_dim) - 1;
    out = at::empty(sizes, src.options());
  }

  at::optional<at::Tensor> arg_out = at::nullopt;
  int64_t *arg_out_data = nullptr;
  if (reduce == "min" || reduce == "max") {
    arg_out = at::full_like(out, src.size(reduce_dim), indptr.options());
    arg_out_data = arg_out.value().DATA_PTR<int64_t>();
  }

  auto N = out.size(reduce_dim) * (indptr.numel() / indptr.size(-1));
  auto K = out.numel() / N;
  auto E = src.size(reduce_dim);

  auto indptr_info = at::cuda::detail::getTensorInfo<int64_t, int>(indptr);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_csr_kernel", [&] {
    auto src_data = src.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      if (K == 1) {
        segment_csr_kernel<scalar_t, REDUCE, 1>
            <<<BLOCKS(32, N), THREADS, 0, stream>>>(
                src_data, indptr_info, out_data, arg_out_data, N, E);
      } else {
        segment_csr_broadcast_kernel<scalar_t, REDUCE>
            <<<BLOCKS(1, N * K), THREADS, 0, stream>>>(
                src_data, indptr_info, out_data, arg_out_data, N, K, E);
      }
    });
  });

  return std::make_tuple(out, arg_out);
}

template <typename scalar_t, ReductionType REDUCE>
__global__ void
segment_coo_kernel(const scalar_t *src_data,
                   const at::cuda::detail::TensorInfo<int64_t, int> index_info,
                   scalar_t *out_data, int64_t *arg_out_data, size_t E) {

  // Each thread processes exactly one entry. Within a warp, we perform a
  // parallel reduction across equal indices, and write the intermediate
  // result via atomics.

  int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int lane_idx = row_idx & (32 - 1);

  if (row_idx < E) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        row_idx, index_info);
    int idx = index_info.data[offset], next_idx;

    scalar_t val = src_data[row_idx], tmp;
    int64_t arg = row_idx % index_info.sizes[index_info.dims - 1], arg_tmp;

#pragma unroll
    for (int i = 1; i < 32; i *= 2) {
      // Parallel reduction inside a single warp.
      tmp = __shfl_up_sync(FULL_MASK, val, i);
      if (REDUCE == MIN || REDUCE == MAX) {
        arg_tmp = __shfl_up_sync(FULL_MASK, arg, i);
      }
      next_idx = __shfl_up_sync(FULL_MASK, idx, i);
      assert(idx >= next_idx);
      if (lane_idx >= i && idx == next_idx)
        Reducer<scalar_t, REDUCE>::update(&val, tmp, &arg, arg_tmp);
    }

    next_idx = __shfl_down_sync(FULL_MASK, idx, 1);
    if (lane_idx == 32 - 1 || idx != next_idx) {
      Reducer<scalar_t, REDUCE>::atom_write(out_data + idx, val,
                                            arg_out_data + idx, arg);
    }
  }
}

template <typename scalar_t, ReductionType REDUCE, int TB>
__global__ void segment_coo_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> index_info,
    scalar_t *out_data, int64_t *arg_out_data, size_t E, size_t K) {

  // Each thread processes a single column and `TB` index entries. Coalesced
  // read and write is performed in column-major order. The intermediate
  // results are written via atomics.

  int row_start = (blockIdx.x * blockDim.y + threadIdx.y) * TB;
  int col_idx = blockIdx.y * blockDim.x + threadIdx.x;

  if (row_start < E && col_idx < K) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        row_start, index_info);

    int idx1 = __ldg(index_info.data + offset);
    scalar_t val = src_data[K * row_start + col_idx];

#pragma unroll
    for (int i = 1; i < TB; i++) {
      if (row_start + i >= E)
        break;

      int idx2 = __ldg(index_info.data + offset +
                       i * index_info.strides[index_info.dims - 1]);
      assert(idx1 <= idx2);
      if (idx1 == idx2) {
        val += src_data[K * (row_start + i) + col_idx];
      } else {
        atomAdd(out_data + K * idx1 + col_idx, val);
        val = src_data[K * (row_start + i) + col_idx];
      }
      idx1 = idx2;
    }

    atomAdd(out_data + K * idx1 + col_idx, val);
  }
}

std::tuple<at::Tensor, at::optional<at::Tensor>>
segment_coo_cuda(at::Tensor src, at::Tensor index, at::Tensor out,
                 std::string reduce) {
  AT_ASSERTM(src.dim() >= index.dim());
  for (int i = 0; i < index.dim(); i++)
    AT_ASSERTM(src.size(i) == index.size(i));

  src = src.contiguous();
  out = out.contiguous();
  auto reduce_dim = index.dim() - 1;

  for (int i = 0; i < out.dim(); i++)
    if (i != reduce_dim)
      AT_ASSERTM(src.size(i) == out.size(i));

  at::optional<at::Tensor> arg_out = at::nullopt;
  if (reduce == "min" || reduce == "max") {
    arg_out = at::full_like(out, src.size(reduce_dim), index.options());
  }

  auto E = index.numel();
  auto K = src.numel() / index.numel();
  auto avg_len = (float)src.size(reduce_dim) / (float)out.size(reduce_dim);

  auto index_info = at::cuda::detail::getTensorInfo<int64_t, int>(index);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_coo_kernel", [&] {
    auto src_data = src.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    // Select the right kernel based on average row length (purely heuristic)
    // and whether we need broadcasting capabilties (K > 1):

    if (K == 1 && reduce == "add") {
      segment_coo_kernel<scalar_t, ADD><<<BLOCKS(1, E), THREADS, 0, stream>>>(
          src_data, index_info, out_data, nullptr, E);
    } else if (K == 1 && reduce == "mean") {
      segment_coo_kernel<scalar_t, MEAN><<<BLOCKS(1, E), THREADS, 0, stream>>>(
          src_data, index_info, out_data, nullptr, E);
    } else if (K == 1 && reduce == "min") {
      auto arg_out_data = arg_out.value().DATA_PTR<int64_t>();
      segment_coo_kernel<scalar_t, MIN><<<BLOCKS(1, E), THREADS, 0, stream>>>(
          src_data, index_info, out_data, arg_out_data, E);
    } else if (K == 1 && reduce == "max") {
      auto arg_out_data = arg_out.value().DATA_PTR<int64_t>();
      segment_coo_kernel<scalar_t, MAX><<<BLOCKS(1, E), THREADS, 0, stream>>>(
          src_data, index_info, out_data, arg_out_data, E);
    } else if (avg_len <= 8)
      segment_coo_broadcast_kernel<scalar_t, ADD, 4>
          <<<dim3(((E + (8 * 4) - 1) / (8 * 4)), (K + 31) / 32), dim3(32, 8), 0,
             stream>>>(src_data, index_info, out_data, nullptr, E, K);
    else if (avg_len <= 16)
      segment_coo_broadcast_kernel<scalar_t, ADD, 8>
          <<<dim3(((E + (8 * 8) - 1) / (8 * 8)), (K + 31) / 32), dim3(32, 8), 0,
             stream>>>(src_data, index_info, out_data, nullptr, E, K);
    else if (avg_len <= 32)
      segment_coo_broadcast_kernel<scalar_t, ADD, 16>
          <<<dim3(((E + (8 * 16) - 1) / (8 * 16)), (K + 31) / 32), dim3(32, 8),
             0, stream>>>(src_data, index_info, out_data, nullptr, E, K);
    else
      segment_coo_broadcast_kernel<scalar_t, ADD, 32>
          <<<dim3(((E + (8 * 32) - 1) / (8 * 32)), (K + 31) / 32), dim3(32, 8),
             0, stream>>>(src_data, index_info, out_data, nullptr, E, K);
  });

  if (reduce == "mean") {
    AT_ASSERTM(false); // TODO: DIVIDE ENTRIES.
  }

  return std::make_tuple(out, arg_out);
}
