#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>

#include "atomics.cuh"
#include "compat.cuh"

#define THREADS 256
#define BLOCKS(TB, N) (TB * N + THREADS - 1) / THREADS
#define FULL_MASK 0xffffffff

// We need our own `IndexToOffset` implementation since we do not want to access
// the last element of the `indexptr`.
template <typename T, typename I> struct IndexPtrToOffset {
  static __host__ __device__ I
  get(I idx, const at::cuda::detail::TensorInfo<T, I> &info) {
    I offset = idx % (info.sizes[info.dims - 1] - 1);
    offset *= info.strides[info.dims - 1];
    idx /= info.sizes[info.dims - 1] - 1;
    for (int i = info.dims - 2; i >= 0; --i) {
      offset += (idx % info.sizes[i]) * info.strides[i];
      idx /= info.sizes[i];
    }
    return offset;
  }
};

template <typename scalar_t, int TB>
__global__ void segment_add_csr_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
    scalar_t *out_data, size_t N, size_t E) {

  // Each warp processes exactly `32/TB` rows. We usually set `TB=32` and only
  // make use of it in case the average row length is less than 32.

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / TB;
  int lane_idx = thread_idx & (TB - 1);

  if (row_idx < N) {
    int offset = IndexPtrToOffset<int64_t, int>::get(row_idx, indptr_info);
    int row_start = __ldg(indptr_info.data + offset);
    int row_end = __ldg(indptr_info.data + offset +
                        indptr_info.strides[indptr_info.dims - 1]);
    scalar_t val = (scalar_t)0;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E;
    for (int src_idx = row_start + lane_idx; src_idx < row_end; src_idx += TB) {
      val += src_data[offset + src_idx]; // "Mostly" coalesced read.
    }

#pragma unroll
    for (int i = TB / 2; i > 0; i /= 2) {
      // Parallel reduction inside a single warp.
      val += __shfl_down_sync(FULL_MASK, val, i);
    }

    if (lane_idx == 0) {
      out_data[row_idx] = val; // "Mostly" coalesced write.
    }
  }
}

template <typename scalar_t>
__global__ void segment_add_csr_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
    scalar_t *out_data, size_t N, size_t K, size_t E) {

  // Each thread processes exactly one row. It turned out that is more efficient
  // than using shared memory due to avoiding synchronization barriers.

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / K;
  int lane_idx = thread_idx % K;

  if (thread_idx < N * K) {
    int offset = IndexPtrToOffset<int64_t, int>::get(row_idx, indptr_info);
    int row_start = __ldg(indptr_info.data + offset);
    int row_end = __ldg(indptr_info.data + offset +
                        indptr_info.strides[indptr_info.dims - 1]);
    scalar_t val = (scalar_t)0;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E * K;
    for (int src_idx = row_start; src_idx < row_end; src_idx++) {
      val += src_data[offset + K * src_idx + lane_idx]; // Coalesced read.
    }

    out_data[thread_idx] = val; // Coalesced write.
  }
}

at::Tensor segment_add_csr_cuda(at::Tensor src, at::Tensor indptr) {
  AT_ASSERTM(src.dim() >= indptr.dim());
  for (int i = 0; i < indptr.dim() - 1; i++)
    AT_ASSERTM(src.size(i) == indptr.size(i));

  src = src.contiguous();

  auto reduce_dim = indptr.dim() - 1;
  auto sizes = src.sizes().vec();
  sizes[reduce_dim] = indptr.size(reduce_dim) - 1;
  auto out = at::empty(sizes, src.options());

  auto N = out.size(reduce_dim) * (indptr.numel() / indptr.size(-1));
  auto K = out.numel() / N;
  auto E = src.size(reduce_dim);
  auto avg_length = (float)src.size(reduce_dim) / (float)out.size(reduce_dim);

  auto indptr_info = at::cuda::detail::getTensorInfo<int64_t, int>(indptr);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_csr_kernel", [&] {
    auto src_data = src.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    // Select the right kernel based on average row length and whether we need
    // broadcasting capabilties (K > 1):
    if (K == 1 && avg_length <= 4) {
      segment_add_csr_kernel<scalar_t, 4><<<BLOCKS(4, N), THREADS, 0, stream>>>(
          src_data, indptr_info, out_data, N, E);
    } else if (K == 1 && avg_length <= 8) {
      segment_add_csr_kernel<scalar_t, 8><<<BLOCKS(8, N), THREADS, 0, stream>>>(
          src_data, indptr_info, out_data, N, E);
    } else if (K == 1 && avg_length <= 16) {
      segment_add_csr_kernel<scalar_t, 16>
          <<<BLOCKS(16, N), THREADS, 0, stream>>>(src_data, indptr_info,
                                                  out_data, N, E);
    } else if (K == 1) {
      segment_add_csr_kernel<scalar_t, 32>
          <<<BLOCKS(32, N), THREADS, 0, stream>>>(src_data, indptr_info,
                                                  out_data, N, E);
    } else {
      segment_add_csr_broadcast_kernel<scalar_t>
          <<<BLOCKS(1, N * K), THREADS, 0, stream>>>(src_data, indptr_info,
                                                     out_data, N, K, E);
    }
  });

  return out;
}

template <typename scalar_t>
__global__ void segment_add_coo_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> index_info,
    scalar_t *out_data, size_t E) {

  // Each thread processes exactly one entry. Within a warp, we perform a
  // parallel reduction across equal indices, and write the intermediate
  // result via atomics.

  int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int lane_idx = row_idx & (32 - 1);

  if (row_idx < E) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        row_idx, index_info);
    int idx = index_info.data[offset], next_idx;
    scalar_t val = src_data[row_idx], tmp;

#pragma unroll
    for (int i = 1; i < 32; i *= 2) {
      tmp = __shfl_up_sync(FULL_MASK, val, i);
      next_idx = __shfl_up_sync(FULL_MASK, idx, i);
      if (lane_idx >= i && idx == next_idx)
        val += tmp;
    }

    next_idx = __shfl_down_sync(FULL_MASK, idx, 1);
    if (lane_idx == 32 - 1 || idx != next_idx) {
      atomAdd(out_data + idx, val);
    }
  }
}

template <typename scalar_t, int TB>
__global__ void segment_add_coo_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> index_info,
    scalar_t *out_data, size_t E, size_t K) {

  // Each thread processes a single column and `TB` rows. Coalesced read and
  // write is performed in column-major order. The intermediate results are
  // written via atomics.

  int row_start = (blockIdx.x * blockDim.y + threadIdx.y) * TB;
  int col_idx = blockIdx.y * blockDim.x + threadIdx.x;

  if (row_start < E && col_idx < K) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        row_start, index_info);

    int idx1 = __ldg(index_info.data + offset);
    scalar_t val = src_data[K * row_start + col_idx];

#pragma unroll
    for (int i = 1; i < TB; i++) {
      if (row_start + i >= E)
        break;

      int idx2 = __ldg(index_info.data + offset +
                       i * index_info.strides[index_info.dims - 1]);
      if (idx1 == idx2) {
        val += src_data[K * (row_start + i) + col_idx];
      } else {
        atomAdd(out_data + K * idx1 + col_idx, val);
        val = src_data[K * (row_start + i) + col_idx];
      }
      idx1 = idx2;
    }

    atomAdd(out_data + K * idx1 + col_idx, val);
  }
}

void segment_add_coo_cuda(at::Tensor src, at::Tensor index, at::Tensor out) {
  AT_ASSERTM(src.dim() >= index.dim());
  for (int i = 0; i < index.dim(); i++)
    AT_ASSERTM(src.size(i) == index.size(i));

  src = src.contiguous();
  auto reduce_dim = index.dim() - 1;

  for (int i = 0; i < out.dim(); i++)
    if (i != reduce_dim)
      AT_ASSERTM(src.size(i) == out.size(i));

  auto E = index.numel();
  auto K = src.numel() / index.numel();
  auto avg_length = (float)src.size(reduce_dim) / (float)out.size(reduce_dim);

  auto index_info = at::cuda::detail::getTensorInfo<int64_t, int>(index);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_add_coo_kernel", [&] {
    auto src_data = src.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    if (K == 1)
      segment_add_coo_kernel<scalar_t><<<BLOCKS(1, E), THREADS, 0, stream>>>(
          src_data, index_info, out_data, E);
    else if (avg_length <= 8)
      segment_add_coo_broadcast_kernel<scalar_t, 4>
          <<<dim3(((E + (8 * 4) - 1) / (8 * 4)), (K + 31) / 32), dim3(32, 8), 0,
             stream>>>(src_data, index_info, out_data, E, K);
    else if (avg_length <= 16)
      segment_add_coo_broadcast_kernel<scalar_t, 8>
          <<<dim3(((E + (8 * 8) - 1) / (8 * 8)), (K + 31) / 32), dim3(32, 8), 0,
             stream>>>(src_data, index_info, out_data, E, K);
    else if (avg_length <= 32)
      segment_add_coo_broadcast_kernel<scalar_t, 16>
          <<<dim3(((E + (8 * 16) - 1) / (8 * 16)), (K + 31) / 32), dim3(32, 8),
             0, stream>>>(src_data, index_info, out_data, E, K);
    else
      segment_add_coo_broadcast_kernel<scalar_t, 32>
          <<<dim3(((E + (8 * 32) - 1) / (8 * 32)), (K + 31) / 32), dim3(32, 8),
             0, stream>>>(src_data, index_info, out_data, E, K);
  });
}
