#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>

#include <THC/THCGeneral.h>
#include <THC/THCThrustAllocator.cuh>

#include <thrust/execution_policy.h>

#include "atomics.cuh"
#include "compat.cuh"
#include "index.cuh"

#define THREADS 256
#define BLOCKS(TB, N) (TB * N + THREADS - 1) / THREADS
#define FULL_MASK 0xffffffff

template <typename T, typename I> struct IndexPtrToOffset {
  static __host__ __device__ I
  get(I idx, const at::cuda::detail::TensorInfo<T, I> &info) {
    I offset = idx % (info.sizes[info.dims - 1] - 1);
    offset *= info.strides[info.dims - 1];
    idx /= info.sizes[info.dims - 1] - 1;
    for (int i = info.dims - 2; i >= 0; --i) {
      offset += (idx % info.sizes[i]) * info.strides[i];
      idx /= info.sizes[i];
    }
    return offset;
  }
};

template <typename scalar_t, int TB>
__global__ void segment_add_csr_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
    scalar_t *out_data, size_t N, size_t E) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / TB;
  int lane_idx = thread_idx & (TB - 1);

  if (row_idx < N) {
    auto offset = IndexPtrToOffset<int64_t, int>::get(row_idx, indptr_info);
    int row_start = __ldg(indptr_info.data + offset);
    int row_end = __ldg(indptr_info.data + offset +
                        indptr_info.strides[indptr_info.dims - 1]);
    scalar_t val = (scalar_t)0;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E;
    for (int src_idx = row_start + lane_idx; src_idx < row_end; src_idx += TB) {
      val += src_data[offset + src_idx];
    }

#pragma unroll
    for (int i = TB / 2; i > 0; i /= 2)
      val += __shfl_down_sync(FULL_MASK, val, i); // Parallel reduction

    if (lane_idx == 0) {
      out_data[row_idx] = val;
    }
  }
}

template <typename scalar_t>
__global__ void segment_add_csr_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
    scalar_t *out_data, size_t N, size_t K, size_t E) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / K;
  int lane_idx = thread_idx % K;

  if (thread_idx < N * K) {
    auto offset = IndexPtrToOffset<int64_t, int>::get(row_idx, indptr_info);
    int row_start = __ldg(indptr_info.data + offset);
    int row_end = __ldg(indptr_info.data + offset +
                        indptr_info.strides[indptr_info.dims - 1]);
    scalar_t val = (scalar_t)0;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E * K;
    for (int src_idx = row_start; src_idx < row_end; src_idx++) {
      // Coalesced read into `src_data`.
      val += src_data[offset + K * src_idx + lane_idx];
    }

    out_data[thread_idx] = val; // Coalesced write into `out_data`
  }
}

at::Tensor segment_add_csr_cuda(at::Tensor src, at::Tensor indptr) {
  AT_ASSERTM(src.dim() >= indptr.dim());
  for (int i = 0; i < indptr.dim() - 1; i++)
    AT_ASSERTM(src.size(i) == indptr.size(i));

  src = src.contiguous();

  auto reduce_dim = indptr.dim() - 1;
  auto sizes = src.sizes().vec();
  sizes[reduce_dim] = indptr.size(reduce_dim) - 1;
  auto out = at::empty(sizes, src.options());

  auto N = out.size(reduce_dim) * (indptr.numel() / indptr.size(-1));
  auto K = out.numel() / N;
  auto E = src.size(reduce_dim);
  auto avg_length = (float)src.size(reduce_dim) / (float)out.size(reduce_dim);

  auto indptr_info = at::cuda::detail::getTensorInfo<int64_t, int>(indptr);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_add_csr_kernel", [&] {
    auto src_data = src.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    if (K == 1 && avg_length <= 4) {
      segment_add_csr_kernel<scalar_t, 4><<<BLOCKS(4, N), THREADS, 0, stream>>>(
          src_data, indptr_info, out_data, N, E);
    } else if (K == 1 && avg_length <= 8) {
      segment_add_csr_kernel<scalar_t, 8><<<BLOCKS(8, N), THREADS, 0, stream>>>(
          src_data, indptr_info, out_data, N, E);
    } else if (K == 1 && avg_length <= 16) {
      segment_add_csr_kernel<scalar_t, 16>
          <<<BLOCKS(16, N), THREADS, 0, stream>>>(src_data, indptr_info,
                                                  out_data, N, E);
    } else if (K == 1) {
      segment_add_csr_kernel<scalar_t, 32>
          <<<BLOCKS(32, N), THREADS, 0, stream>>>(src_data, indptr_info,
                                                  out_data, N, E);
    } else {
      segment_add_csr_broadcast_kernel<scalar_t>
          <<<BLOCKS(1, N * K), THREADS, 0, stream>>>(src_data, indptr_info,
                                                     out_data, N, K, E);
    }
  });

  return out;
}

template <typename scalar_t, int TB>
__global__ void segment_add_coo_kernel(const scalar_t *src_data,
                                       const int64_t *index_data,
                                       scalar_t *out_data, size_t numel) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int lane_idx = thread_idx & (TB - 1);

  if (thread_idx < numel) {
    auto idx = __ldg(index_data + thread_idx);
    scalar_t val = src_data[thread_idx], tmp;

#pragma unroll
    for (int offset = 1; offset < TB; offset *= 2) {
      tmp = __shfl_up_sync(FULL_MASK, val, offset);
      if (lane_idx >= offset &&
          idx == __ldg(index_data + thread_idx - offset)) {
        val += tmp;
      }
    }

    if (lane_idx == TB - 1 || idx != __ldg(index_data + thread_idx + 1)) {
      atomAdd(out_data + idx, val);
    }
  }
}

void segment_add_coo_cuda(at::Tensor src, at::Tensor index, at::Tensor out) {
  auto numel = src.numel();
  auto avg_length = (float)numel / (float)out.numel();

  auto index_data = index.DATA_PTR<int64_t>();
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_add_coo_kernel", [&] {
    auto src_data = src.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    segment_add_coo_kernel<scalar_t, 32>
        <<<BLOCKS(1, numel), THREADS, 0, stream>>>(src_data, index_data,
                                                   out_data, numel);
  });
}

void segment_add_thrust_cuda(at::Tensor src, at::Tensor index, at::Tensor out) {
  auto stream = at::cuda::getCurrentCUDAStream();
  auto allocator = THCThrustAllocator(at::globalContext().lazyInitCUDA());
  auto policy = thrust::cuda::par(allocator).on(stream);

  auto key = at::full_like(out, -1, out.options().dtype(at::kLong));

  auto index_data = thrust::device_ptr<int64_t>(index.DATA_PTR<int64_t>());
  auto key_data = thrust::device_ptr<int64_t>(key.DATA_PTR<int64_t>());

  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_add_thrust_kernel", [&] {
    auto src_data = thrust::device_ptr<scalar_t>(src.DATA_PTR<scalar_t>());
    auto out_data = thrust::device_ptr<scalar_t>(out.DATA_PTR<scalar_t>());

    thrust::reduce_by_key(policy, index_data, index_data + index.numel(),
                          src_data, key_data, out_data);
  });
}
