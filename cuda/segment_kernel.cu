#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <torch/extension.h>

#include "atomics.cuh"
#include "compat.cuh"
#include "indptr.cuh"

#define THREADS 256
#define BLOCKS(TB, N) (TB * N + THREADS - 1) / THREADS
#define FULL_MASK 0xffffffff

enum ReductionType { SUM, MEAN, MIN, MAX };

const std::map<std::string, ReductionType> reduce2REDUCE = {
    {"sum", SUM}, {"add", SUM}, {"mean", MEAN}, {"min", MIN}, {"max", MAX},
};

#define AT_DISPATCH_REDUCTION_TYPES(reduce, ...)                               \
  [&] {                                                                        \
    switch (reduce2REDUCE.at(reduce)) {                                        \
    case SUM: {                                                                \
      const ReductionType REDUCE = SUM;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MEAN: {                                                               \
      const ReductionType REDUCE = MEAN;                                       \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MIN: {                                                                \
      const ReductionType REDUCE = MIN;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    case MAX: {                                                                \
      const ReductionType REDUCE = MAX;                                        \
      return __VA_ARGS__();                                                    \
    }                                                                          \
    }                                                                          \
  }()

template <typename scalar_t, ReductionType REDUCE> struct Reducer {
  static inline __host__ __device__ scalar_t init() {
    if (REDUCE == MIN) {
      return std::numeric_limits<scalar_t>::max();
    } else if (REDUCE == MAX) {
      return std::numeric_limits<scalar_t>::lowest();
    } else {
      return (scalar_t)0;
    }
  }

  static inline __host__ __device__ void update(scalar_t *val,
                                                scalar_t new_val) {
    if (REDUCE == SUM || REDUCE == MEAN) {
      *val = *val + new_val;
    } else if ((REDUCE == MIN && new_val < *val) ||
               (REDUCE == MAX && new_val > *val)) {
      *val = new_val;
    }
  }

  static inline __host__ __device__ void update(scalar_t *val, scalar_t new_val,
                                                int64_t *arg, int64_t new_arg) {
    if (REDUCE == SUM || REDUCE == MEAN) {
      *val = *val + new_val;
    } else if ((REDUCE == MIN && new_val < *val) ||
               (REDUCE == MAX && new_val > *val)) {
      *val = new_val;
      *arg = new_arg;
    }
  }

  static inline __host__ __device__ void write(scalar_t *address, scalar_t val,
                                               int64_t *arg_address,
                                               int64_t arg, int count) {
    if (REDUCE == SUM) {
      *address = val;
    } else if (REDUCE == MEAN) {
      *address = val / (scalar_t)max(count, 1);
    } else if (REDUCE == MIN || REDUCE == MAX) {
      if (count > 0) {
        *address = val;
        *arg_address = arg;
      } else {
        *address = (scalar_t)0;
      }
    }
  }

  static inline __device__ void atomic_write(scalar_t *address, scalar_t val) {
    if (REDUCE == SUM || REDUCE == MEAN) {
      atomAdd(address, val);
    } else if (REDUCE == MIN && val < *address) {
      atomMin(address, val);
    } else if (REDUCE == MAX && val > *address) {
      atomMax(address, val);
    }
  }
};

template <typename scalar_t, ReductionType REDUCE, int TB>
__global__ void
segment_csr_kernel(const scalar_t *src_data,
                   const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
                   scalar_t *out_data, int64_t *arg_out_data, size_t N,
                   size_t E) {

  // Each warp processes exactly `32/TB` rows and aggregates all row values
  // via a parallel reduction.

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / TB;
  int lane_idx = thread_idx & (TB - 1);

  if (row_idx < N) {
    int offset = IndexPtrToOffset<int64_t>::get(row_idx, indptr_info);
    int64_t row_start = __ldg(indptr_info.data + offset);
    int64_t row_end = __ldg(indptr_info.data + offset +
                            indptr_info.strides[indptr_info.dims - 1]);

    scalar_t val = Reducer<scalar_t, REDUCE>::init();
    int64_t arg, arg_tmp;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E;
    for (int64_t src_idx = row_start + lane_idx; src_idx < row_end;
         src_idx += TB) {
      Reducer<scalar_t, REDUCE>::update(&val, src_data[offset + src_idx], &arg,
                                        src_idx);
    }

#pragma unroll
    for (int i = TB / 2; i > 0; i /= 2) {
      // Parallel reduction inside a single warp.
      if (REDUCE == MIN || REDUCE == MAX)
        arg_tmp = __shfl_down_sync(FULL_MASK, arg, i);
      Reducer<scalar_t, REDUCE>::update(
          &val, __shfl_down_sync(FULL_MASK, val, i), &arg, arg_tmp);
    }

    if (lane_idx == 0) {
      Reducer<scalar_t, REDUCE>::write(out_data + row_idx, val,
                                       arg_out_data + row_idx, arg,
                                       row_end - row_start);
    }
  }
}

template <typename scalar_t, ReductionType REDUCE>
__global__ void segment_csr_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
    scalar_t *out_data, int64_t *arg_out_data, size_t N, size_t K, size_t E) {

  // Each thread processes exactly one row. It turned out that is more
  // efficient than using shared memory due to avoiding synchronization
  // barriers.

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / K;
  int lane_idx = thread_idx % K;

  if (thread_idx < N * K) {
    int offset = IndexPtrToOffset<int64_t>::get(row_idx, indptr_info);
    int64_t row_start = __ldg(indptr_info.data + offset);
    int64_t row_end = __ldg(indptr_info.data + offset +
                            indptr_info.strides[indptr_info.dims - 1]);

    scalar_t val = Reducer<scalar_t, REDUCE>::init();
    int64_t arg;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E * K;
    for (int64_t src_idx = row_start; src_idx < row_end; src_idx++) {
      Reducer<scalar_t, REDUCE>::update(
          &val, src_data[offset + K * src_idx + lane_idx], &arg, src_idx);
    }

    Reducer<scalar_t, REDUCE>::write(out_data + thread_idx, val,
                                     arg_out_data + thread_idx, arg,
                                     row_end - row_start);
  }
}

std::tuple<torch::Tensor, torch::optional<torch::Tensor>>
segment_csr_cuda(torch::Tensor src, torch::Tensor indptr,
                 torch::optional<torch::Tensor> out_opt, std::string reduce) {

  hipSetDevice(src.get_device());

  AT_ASSERTM(src.dim() >= indptr.dim(), "Input mismatch");

  // Broadcasting `indptr` via `expand`.
  auto sizes = indptr.sizes().vec();
  for (int i = 0; i < indptr.dim() - 1; i++) {
    sizes[i] = src.size(i);
  }
  indptr = indptr.expand(sizes);

  src = src.contiguous();
  auto reduce_dim = indptr.dim() - 1;

  torch::Tensor out;
  if (out_opt.has_value()) {
    out = out_opt.value().contiguous();
    for (int i = 0; i < out.dim(); i++)
      if (i != reduce_dim)
        AT_ASSERTM(src.size(i) == out.size(i), "Input mismatch");
    AT_ASSERTM(out.size(reduce_dim) == indptr.size(reduce_dim) - 1,
               "Input mismatch");
  } else {
    sizes = src.sizes().vec();
    sizes[reduce_dim] = indptr.size(reduce_dim) - 1;
    out = torch::empty(sizes, src.options());
  }

  torch::optional<torch::Tensor> arg_out = torch::nullopt;
  int64_t *arg_out_data = nullptr;
  if (reduce2REDUCE.at(reduce) == MIN || reduce2REDUCE.at(reduce) == MAX) {
    arg_out = torch::full_like(out, src.size(reduce_dim), indptr.options());
    arg_out_data = arg_out.value().DATA_PTR<int64_t>();
  }

  auto N = out.size(reduce_dim) * (indptr.numel() / indptr.size(-1));
  auto K = out.numel() / N;
  auto E = src.size(reduce_dim);

  auto indptr_info = at::cuda::detail::getTensorInfo<int64_t, int>(indptr);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_csr_kernel", [&] {
    auto src_data = src.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      if (K == 1) {
        segment_csr_kernel<scalar_t, REDUCE, 1>
            <<<BLOCKS(32, N), THREADS, 0, stream>>>(
                src_data, indptr_info, out_data, arg_out_data, N, E);
      } else {
        segment_csr_broadcast_kernel<scalar_t, REDUCE>
            <<<BLOCKS(1, N * K), THREADS, 0, stream>>>(
                src_data, indptr_info, out_data, arg_out_data, N, K, E);
      }
    });
  });

  return std::make_tuple(out, arg_out);
}

template <typename scalar_t, ReductionType REDUCE, bool HAS_VAL>
__global__ void
segment_coo_kernel(const scalar_t *src_data,
                   const at::cuda::detail::TensorInfo<int64_t, int> index_info,
                   scalar_t *out_data, size_t E, size_t N) {

  // Each thread processes exactly one entry. Within a warp, we perform a
  // parallel reduction across equal indices, and write the intermediate
  // result via atomics.

  int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int lane_idx = row_idx & (32 - 1);
  int D = index_info.sizes[index_info.dims - 1];

  if (row_idx < E) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        row_idx, index_info);
    int64_t idx = index_info.data[offset], next_idx;
    int out_idx = (row_idx / D) * N + idx;

    scalar_t val = HAS_VAL ? src_data[row_idx] : (scalar_t)1, tmp;

#pragma unroll
    for (int i = 1; i < 32; i *= 2) {
      // Parallel reduction inside a single warp.
      tmp = __shfl_up_sync(FULL_MASK, val, i);
      next_idx = __shfl_up_sync(FULL_MASK, idx, i);
      if (lane_idx >= i && row_idx / D == (row_idx - i) / D) {
        assert(idx >= next_idx);
        if (idx == next_idx)
          Reducer<scalar_t, REDUCE>::update(&val, tmp);
      }
    }

    next_idx = __shfl_down_sync(FULL_MASK, idx, 1);
    if (lane_idx == 32 - 1 || row_idx / D != (row_idx + 1) / D ||
        idx != next_idx)
      Reducer<scalar_t, REDUCE>::atomic_write(out_data + out_idx, val);
  }
}

template <typename scalar_t>
__global__ void segment_coo_arg_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> index_info,
    scalar_t *out_data, int64_t *arg_out_data, size_t E, size_t N) {

  int row_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int D = index_info.sizes[index_info.dims - 1];

  if (row_idx < E) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        row_idx, index_info);
    int64_t idx = index_info.data[offset];
    int out_idx = (row_idx / D) * N + idx;

    scalar_t val = __ldg(out_data + out_idx);
    if (src_data[row_idx] == val)
      arg_out_data[out_idx] = row_idx % D;
  }
}

template <typename scalar_t, ReductionType REDUCE, int TB>
__global__ void segment_coo_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> index_info,
    scalar_t *out_data, size_t E, size_t K, size_t N) {

  // Each thread processes a single column and `TB` index entries. Coalesced
  // read and write is performed in column-major order. The intermediate
  // results are written via atomics.

  int D = index_info.sizes[index_info.dims - 1];
  int E_1 = E / D;
  int E_2 = D + TB - (D % TB);

  int row_idx = blockIdx.x * blockDim.y + threadIdx.y;
  int col_idx = blockIdx.y * blockDim.x + threadIdx.x;

  int dim_start = (row_idx * TB) / E_2;
  int row_start = (row_idx * TB) % E_2;

  if (dim_start < E_1 && col_idx < K) {

    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        dim_start * D + row_start, index_info);
    int idx1 = __ldg(index_info.data + offset), idx2;

    scalar_t val = src_data[K * (dim_start * D + row_start) + col_idx];

#pragma unroll
    for (int i = 1; i < TB; i++) {
      if (row_start + i >= D)
        break;

      idx2 = __ldg(index_info.data + offset +
                   i * index_info.strides[index_info.dims - 1]);
      assert(idx1 <= idx2);
      if (idx1 == idx2) {
        Reducer<scalar_t, REDUCE>::update(
            &val, src_data[K * (dim_start * D + row_start + i) + col_idx]);
      } else {
        Reducer<scalar_t, REDUCE>::atomic_write(
            out_data + (dim_start * N + idx1) * K + col_idx, val);
        val = src_data[K * (dim_start * D + row_start + i) + col_idx];
      }

      idx1 = idx2;
    }

    Reducer<scalar_t, REDUCE>::atomic_write(
        out_data + (dim_start * N + idx1) * K + col_idx, val);
  }
}

template <typename scalar_t>
__global__ void segment_coo_arg_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> index_info,
    scalar_t *out_data, int64_t *arg_out_data, size_t E, size_t K, size_t N) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / K;
  int col_idx = thread_idx % K;
  int D = index_info.sizes[index_info.dims - 1];

  if (row_idx < E && col_idx < K) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        row_idx, index_info);
    int idx = __ldg(index_info.data + offset);
    int out_idx = ((row_idx / D) * N + idx) * K + col_idx;

    scalar_t val = __ldg(out_data + out_idx);
    if (src_data[thread_idx] == val)
      arg_out_data[out_idx] = row_idx % D;
  }
}

std::tuple<torch::Tensor, torch::optional<torch::Tensor>>
segment_coo_cuda(torch::Tensor src, torch::Tensor index, torch::Tensor out,
                 std::string reduce) {

  hipSetDevice(src.get_device());

  AT_ASSERTM(src.dim() >= index.dim(), "Input mismatch");

  // Broadcasting `index` via `expand`.
  auto sizes = index.sizes().vec();
  for (int i = 0; i < index.dim(); i++) {
    sizes[i] = src.size(i);
  }
  index = index.expand(sizes);

  src = src.contiguous();
  out = out.contiguous();
  auto reduce_dim = index.dim() - 1;

  for (int i = 0; i < out.dim(); i++)
    if (i != reduce_dim)
      AT_ASSERTM(src.size(i) == out.size(i), "Input mismatch");

  torch::optional<torch::Tensor> arg_out = torch::nullopt;
  int64_t *arg_out_data = nullptr;
  if (reduce2REDUCE.at(reduce) == MIN || reduce2REDUCE.at(reduce) == MAX) {
    arg_out = torch::full_like(out, src.size(reduce_dim), index.options());
    arg_out_data = arg_out.value().DATA_PTR<int64_t>();
  }

  auto E = index.numel();
  auto E_2 = index.size(reduce_dim);
  auto E_1 = index.numel() / E_2;
  auto K = src.numel() / E;
  auto N = out.size(reduce_dim);
  auto avg_len = (float)E_2 / (float)N;

  auto index_info = at::cuda::detail::getTensorInfo<int64_t, int>(index);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES(src.scalar_type(), "segment_coo_kernel", [&] {
    auto src_data = src.DATA_PTR<scalar_t>();
    auto out_data = out.DATA_PTR<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      if (K == 1) {
        segment_coo_kernel<scalar_t, REDUCE, true>
            <<<BLOCKS(1, E), THREADS, 0, stream>>>(src_data, index_info,
                                                   out_data, E, N);
      } else if (avg_len <= 8) {
        segment_coo_broadcast_kernel<scalar_t, REDUCE, 4>
            <<<dim3((E_1 * ((E_2 + 3) / 4) + 7) / 8, (K + 31) / 32),
               dim3(32, 8), 0, stream>>>(src_data, index_info, out_data, E, K,
                                         N);
      } else if (avg_len <= 16) {
        segment_coo_broadcast_kernel<scalar_t, REDUCE, 8>
            <<<dim3((E_1 * ((E_2 + 7) / 8) + 7) / 8, (K + 31) / 32),
               dim3(32, 8), 0, stream>>>(src_data, index_info, out_data, E, K,
                                         N);
      } else if (avg_len <= 32) {
        segment_coo_broadcast_kernel<scalar_t, REDUCE, 16>
            <<<dim3((E_1 * ((E_2 + 15) / 16) + 7) / 8, (K + 31) / 32),
               dim3(32, 8), 0, stream>>>(src_data, index_info, out_data, E, K,
                                         N);
      } else {
        segment_coo_broadcast_kernel<scalar_t, REDUCE, 32>
            <<<dim3((E_1 * ((E_2 + 31) / 32) + 7) / 8, (K + 31) / 32),
               dim3(32, 8), 0, stream>>>(src_data, index_info, out_data, E, K,
                                         N);
      }

      if (REDUCE == MIN || REDUCE == MAX) {
        if (K == 1) {
          segment_coo_arg_kernel<scalar_t>
              <<<BLOCKS(1, E), THREADS, 0, stream>>>(
                  src_data, index_info, out_data, arg_out_data, E, N);
        } else {
          segment_coo_arg_broadcast_kernel<scalar_t>
              <<<BLOCKS(1, E * K), THREADS, 0, stream>>>(
                  src_data, index_info, out_data, arg_out_data, E, K, N);
        }
      }
    });
  });

  if (reduce2REDUCE.at(reduce) == MEAN) {
    auto sizes = index.sizes().vec();
    sizes[reduce_dim] = out.size(reduce_dim);
    auto count = torch::zeros(sizes, out.options());

    AT_DISPATCH_ALL_TYPES(out.scalar_type(), "count_kernel", [&] {
      auto count_data = count.DATA_PTR<scalar_t>();
      segment_coo_kernel<scalar_t, SUM, false>
          <<<BLOCKS(1, E), THREADS, 0, stream>>>(nullptr, index_info,
                                                 count_data, E, N);
    });

    count.clamp_(1);
    arg_out = count;

    for (int i = reduce_dim + 1; i < out.dim(); i++) {
      count = count.unsqueeze(-1);
    }

    out.div_(count);
  }

  return std::make_tuple(out, arg_out);
}
